/*
============================================================================
Filename    : algorithm.c
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}


__global__
void gpu_calculation(double* input, double* output, int length)
{   
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int index = y * length + x;
    
    if(x == length / 2 - 1 && y == length / 2 - 1) {
        return;
    }
    if(x == length / 2 && y == length / 2 - 1) {
        return;
    }
    if(x == length / 2 - 1 && y == length / 2) {
        return;
    }
    if(x == length / 2 && y == length / 2) {
        return;
    }

    if( (x > 0) && (x < length - 1) && (y > 0) && (y < length - 1) ) 
    {
        output[index] = (input[(x-1)*(length)+(y-1)] +
                        input[(x-1)*(length)+(y)]   +
                        input[(x-1)*(length)+(y+1)] +
                        input[(x)*(length)+(y-1)]   +
                        input[(x)*(length)+(y)]     +
                        input[(x)*(length)+(y+1)]   +
                        input[(x+1)*(length)+(y-1)] +
                        input[(x+1)*(length)+(y)]   +
                        input[(x+1)*(length)+(y+1)]) / 9;
            
    }
    
}

// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    double* gpu_input;
    double* gpu_output;
    size_t size = length*length*sizeof(double);
    dim3 nbrThreads(8,8);
    dim3 nbrBlocks(length/8+1, length/8+1);
    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
    hipMalloc((void**)&gpu_input, size);
    hipMalloc((void**)&gpu_output, size);
    hipMemcpy((void*)gpu_input, (void*)input,size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    hipMemcpy((void*)gpu_output, (void*)output,size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    //Copy array from host to device
    hipEventRecord(comp_start);
    /* GPU calculation goes here */
    for(int i = 0; i < iterations-1; i++)
    {
        gpu_calculation<<<nbrBlocks,nbrThreads>>>(gpu_input, gpu_output, length);
        hipDeviceSynchronize();
        double* temp = gpu_output;
        gpu_output = gpu_input;
        gpu_input = temp;

    }
    gpu_calculation<<<nbrBlocks,nbrThreads>>>(gpu_input, gpu_output, length);

    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    hipMemcpy((void*)output, (void*)gpu_output,size, hipMemcpyDeviceToHost);

    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */
    hipFree(gpu_input);
    hipFree(gpu_output);
    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}