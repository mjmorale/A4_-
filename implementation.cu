/*
============================================================================
Filename    : algorithm.c
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}


__global__
void gpu_calculation(double* input, double* output, int length)
{

}

// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    double* gpu_input;
    double* gpu_output;
    int size = length*length*sizeof(double);
    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
    hipMalloc((void**)&gpu_input, size);
    hipMalloc((void**)&gpu_output, size);
    hipMemcpy((void*)gpu_input, (void*)input,size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    hipMemcpy((void*)gpu_output, (void*)output,size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    //Copy array from host to device
    hipEventRecord(comp_start);
    /* GPU calculation goes here */
    for(int i = 0; i < iterations; i++)
    {
        gpu_calculation<<<1024,1024>>>(gpu_input, gpu_output, length);
        hipDeviceSynchronize();
        double* temp = gpu_output;
        gpu_output = gpu_input;
        gpu_input = temp;

    }
    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    hipMemcpy((void*)output, (void*)gpu_output,size, hipMemcpyDeviceToHost);

    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */
    hipFree(gpu_input);
    hipFree(gpu_output);
    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}