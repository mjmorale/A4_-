/*
============================================================================
Filename    : algorithm.c
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}


__global__
void gpu_calculation(double* input, double* output, int length)
{   
    printf("1");
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    printf("2");
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    printf("3");
    unsigned int index = y * length + x;
    printf("4");

    int b = x> 1;
    int c = x < length-1;
    int d = y < length-1;
    int e = y > 1;

    printf("b%d c%d d%d e%d f", &b, &c, &d, &e);

    if( (x > 1) && (x < length - 1) && (y > 1) && (y < length - 1) ) {
        printf("6");
        output[index] = 2;/*(input[(x-1)*(length)+(y-1)] +
                        input[(x-1)*(length)+(y)]   +
                        input[(x-1)*(length)+(y+1)] +
                        input[(x)*(length)+(y-1)]   +
                        input[(x)*(length)+(y)]     +
                        input[(x)*(length)+(y+1)]   +
                        input[(x+1)*(length)+(y-1)] +
                        input[(x+1)*(length)+(y)]   +
                        input[(x+1)*(length)+(y+1)]) / 9;*/
            
    }
    printf("5\n");
    if(x == length / 2 - 1 && y == length / 2 - 1) {
        return;
    }
    if(x == length / 2 && y == length / 2 - 1) {
        return;
    }
    if(x == length / 2 - 1 && y == length / 2) {
        return;
    }
    if(x == length / 2 && y == length / 2) {
        return;
    }
}

// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    double* gpu_input;
    double* gpu_output;
    int size = length*length*sizeof(double);
    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
    hipMalloc((void**)&gpu_input, size);
    hipMalloc((void**)&gpu_output, size);
    hipMemcpy((void*)gpu_input, (void*)input,size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    hipMemcpy((void*)gpu_output, (void*)output,size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    //Copy array from host to device
    hipEventRecord(comp_start);
    /* GPU calculation goes here */
    for(int i = 0; i < iterations; i++)
    {
        gpu_calculation<<<1,1>>>(gpu_input, gpu_output, length);
        hipDeviceSynchronize();
        double* temp = gpu_output;
        gpu_output = gpu_input;
        gpu_input = temp;

    }
    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    hipMemcpy((void*)output, (void*)gpu_output,size, hipMemcpyDeviceToHost);

    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */
    hipFree(gpu_input);
    hipFree(gpu_output);
    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}